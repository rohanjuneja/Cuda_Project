#include "hip/hip_runtime.h"
// Find high divergence points of a vector field
// --- Input: 1. normalized 3D vector field
//
//               dFx     dFy     dFz
// divergence = ----- + ----- + -----
//               dx      dy      dz
//
// --- Output: highest ...% divergence point list
// --- Author: Nicu D. Cornea, Vizlab, Rutgers University
// --- Date: Wed Aug 20 17:53:56 EDT 2003
//

#include "HighDiverg.h"
#include <thrust/copy.h>


// #define TRACE

#define SEARCH_GRID		1
#define CELL_SIZE		1.00 / SEARCH_GRID

#define MAX_NUM_HDPTS	5000

/*
this structure will be used in the buffer for storing the divergence values
*/
typedef struct Divergence_vals
{
  /* data */
  bool flag;
  double val;
  Divergence_vals()
  {
    flag=false;
  }

}Divergence_vals;
      
struct find_div_pts
{
  __host__ __device__
  bool operator()(const Divergence_vals x)
  {
    return x.flag;
  }
};



typedef struct {
	int* Points;
	int numPoints;
} HDGroup;
// Lock mylock=new Lock();

inline bool PointIsCloseToGroup(int pt, int grp, HDGroup *Groups, VoxelPositionDouble **HDPts);

__host__ __device__ Vector interpolation(double x, double y, double z, int sizx, int sizy, int sizz, Vector *forcevec);
/*
this kernel calculates the divergence value at each voxel with the help of force field
and saves in the custom structure buffer 
*/
__global__ void max_min_divergence(unsigned char *flags,Vector *ForceField,double *maxDiv,double *minDiv,bool inOut,int slsz,int L,int M, int N, double *vdist,Divergence_vals *d_arr)
{
  int k=blockIdx.x+1;
  int j=blockIdx.y+1;
  int i=threadIdx.x+1;
  double div;
  int idx=k*slsz + j*L +i;
  d_arr[idx].flag = false;

  if(!inOut) {
    // - if this point is EXTERIOR, BOUNDARY or SURF, skip it
    if( (flags[idx] == EXTERIOR) ||
    (flags[idx] == BOUNDARY) ||
    (flags[idx] == SURF))
    {
      return;
    }
  }
  else {
    // we look for high divergence points outside the object too
    // ignore only boundary points.
    if( (flags[idx] == BOUNDARY) ||
        (flags[idx] == SURF))
    {
      return;
    }
  }

  double x, y, z;
  for(int kk=0; kk < SEARCH_GRID; kk++) {
    for(int jj=0; jj < SEARCH_GRID; jj++) {
      for(int ii=0; ii < SEARCH_GRID; ii++) {
        x = i + (ii * CELL_SIZE);
        y = j + (jj * CELL_SIZE);
        z = k + (kk * CELL_SIZE);
#ifdef TRACE
        //              printf("At point: (%lf, %lf, %lf)\n", x, y, z);
#endif              
        // interpolate force vectors arround the point
        
        Vector v_0 = interpolation(x + *vdist, y, z, L, M, N, ForceField);
        Vector v_1 = interpolation(x - *vdist, y, z, L, M, N, ForceField);
        Vector v_2 = interpolation(x, y + *vdist, z, L, M, N, ForceField);
        Vector v_3 = interpolation(x, y - *vdist, z, L, M, N, ForceField);
        Vector v_4 = interpolation(x, y, z + *vdist, L, M, N, ForceField);
        Vector v_5 = interpolation(x, y, z - *vdist, L, M, N, ForceField);
        
        div = ((v_0.xd - v_1.xd) + (v_2.yd - v_3.yd) + (v_4.zd - v_5.zd)) / (2 * *vdist);
        d_arr[idx].flag=true;
        d_arr[idx].val=div;
      }
    }
  }



}

// double GetDiv(double x, double y, double z);

bool GetHighDivergencePoints(
	Vector* ForceField, 	      // [in] vector field
	int L, int M, int N,	      // [in] size of vector field (X, Y and Z)
	unsigned char *flags,	      // [in] flags array
	float perc,		      // [in] percentage of high div. points 
	                              //         to be returned (top <perc> %)
	VoxelPositionDouble **HDPts,  // [out] high divergence point list
	int *numHDPts,		      // [out] number of points in the list
	bool inOut                    // [in] flag specifying if we should look
	                              //    outside the object too (if true).
	                              // DEFAULT: false
) {

#ifdef TRACE
  printf("TRACE: Starting GetHighDivergencePoints function. Cellsize = %lf\n", CELL_SIZE);
#endif

  (*HDPts) = NULL;
  (*numHDPts) = 0;

  if(perc == 0) {
    return true;
  }
  
  long idx, slsz;
  int i,j,k, ii, jj, kk, s;
  double x, y, z;
  long cntz, cntnz;
  
  slsz = L*M;		// slice size
  double adiv[MAX_NUM_HDPTS];	// divergence array
  
    
  if(((*HDPts) = new VoxelPositionDouble[MAX_NUM_HDPTS]) == NULL) {
    printf("GetHighDivergencePoints: UPS! - Error allocating memory for the output array. Abort.\n");
    exit(1);
  }

  
  // calculate divergence throughout the dataset
  double maxDiv = -999999.99;
  double minDiv =  999999.99;
  double div;
  
  cntz = 0;
  cntnz = 0;
  double zerodiv = 0.1;
  
  /////////////////////////////////////
  Vector v[6];
  double vdist = (CELL_SIZE) / 2.00;
// #ifdef TRACE
  // printf("vdist = %lf\n", vdist);
// #endif	
  
  printf("Finding high divergence points (1).\n");
  unsigned char *d_flags;
  Vector *d_ForceField;
  double *d_maxDiv;
  double *d_minDiv;
  double *d_vdist;
  Divergence_vals *d_arr;
  Divergence_vals *h_arr,*h_result;
  h_arr=(Divergence_vals *)malloc(sizeof(Divergence_vals)*L*M*N);
  h_result=(Divergence_vals *)malloc(sizeof(Divergence_vals)*L*M*N);
  hipMalloc((void **)&d_flags,sizeof(unsigned char)*L*M*N);
  hipMalloc((void **)&d_ForceField,sizeof(Vector)*L*M*N);
  hipMalloc((void **)&d_minDiv,sizeof(double));
  hipMalloc((void **)&d_arr, sizeof(Divergence_vals)*L*M*N);
  hipMalloc((void **)&d_vdist,sizeof(double));

  hipMemcpy(d_flags,flags,sizeof(unsigned char)*L*M*N,hipMemcpyHostToDevice);
  hipMemcpy(d_ForceField,ForceField,sizeof(Vector)*L*M*N,hipMemcpyHostToDevice);
  hipMemcpy(d_maxDiv,&maxDiv,sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_minDiv,&minDiv,sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(d_vdist, &vdist, sizeof(double), hipMemcpyHostToDevice);

  dim3 dimBlock(L-2);
  dim3 dimGrid(N-2,M-2);

  max_min_divergence<<<dimGrid,dimBlock>>>(d_flags,d_ForceField,d_maxDiv,d_minDiv,inOut,slsz,L,M,N,d_vdist,d_arr);
 

  hipMemcpy(h_arr,d_arr,sizeof(Divergence_vals)*L*M*N,hipMemcpyDeviceToHost);
 
  for(k=1;k<N-1;k++){   
    for(j=1;j<M-1;j++){
      for(i=1;i<L-1;i++){
        idx = k*slsz + j*L +i;
        if(h_arr[idx].flag==true){
          if(h_arr[idx].val > maxDiv) {
              // printf("%.6f %.6f\n", maxDiv, h_arr[idx].val);
              maxDiv = h_arr[idx].val;
          }
          if(h_arr[idx].val < minDiv) {
              // printf("%.6f %.6f\n", minDiv, h_arr[idx].val);
              minDiv = h_arr[idx].val ;
          }
        }

      }
    }

  }

// #ifdef _DEBUG
  printf("Divergence: max = %.6f, min = %.6f\n", maxDiv, minDiv);
// #endif
  
  double threshold;
  
  
  // case 1:
  // take <perc> percent of the lowest negative value:
  // !! have to change the comparison
  threshold = maxDiv - minDiv;
  threshold = ((double)perc / 100.00) * threshold;
  threshold = minDiv + threshold;
  printf("Finding high divergence points (2).\n"); 
	for(k=1;k<N-1;k++){   
    for(j=1;j<M-1;j++){
      for(i=1;i<L-1;i++){
        idx = k*slsz + j*L +i;
        if(h_arr[idx].flag==true){
          if(h_arr[idx].val <= threshold) {
            // add the point to the HD list
            (*HDPts)[(*numHDPts)].x = i;
            (*HDPts)[(*numHDPts)].y = j;
            (*HDPts)[(*numHDPts)].z = k;
    
            adiv[(*numHDPts)] = h_arr[idx].val;
    
            (*numHDPts) = (*numHDPts) + 1;
            if((*numHDPts) >= MAX_NUM_HDPTS) {
              printf("UPS! Too many high divergence points detected. \
                   Reached maximum of %d. Abort\n", MAX_NUM_HDPTS);
              exit(1);
            } 
          }      
        }

      }
    }
  }
  
  /*
  // case 2:
  // take <perc> percent of the highest pozitive value:
  // !! have to change the comparison
  // NOT GOOD
  threshold = maxDiv - minDiv;
  threshold = ((double)perc / 100.00) * threshold;
  threshold = maxDiv - threshold;
  */
  /*
  // case 3:
  // take <perc> percent of the lowest value (must be negative):
  // !! have to change the comparison
  // NOT GOOD
  threshold = minDiv;
  threshold = ((double)perc / 100.00) * threshold;
  threshold = minDiv - threshold;
  */

#ifdef _DEBUG
  printf("Threshold set to: %lf\n", threshold);
  printf("Number of close to 0 divergence points [-%lf..%lf]: %ld. \n \
                Number of non 0 divergence points: %ld.\n", 
	 zerodiv, zerodiv, cntz, cntnz);
#endif

  
	
  //
  // sort the points on the divergence value;
  //
  
  double minval, tmp;
  int minpos;
  
  for(i=0; i < (*numHDPts); i++) {
    minval = adiv[i];
    minpos = i;
    for(j=i+1; j < (*numHDPts); j++) {
      if(adiv[j] < minval) {
	minval = adiv[j];
	minpos = j;
      }
    }
    if(minpos != i) {
      // exchange points and div values
      tmp = adiv[i];
      adiv[i] = adiv[minpos];
      adiv[minpos] = tmp;
      
      tmp = (*HDPts)[i].x; (*HDPts)[i].x = (*HDPts)[minpos].x; (*HDPts)[minpos].x = tmp;
      tmp = (*HDPts)[i].y; (*HDPts)[i].y = (*HDPts)[minpos].y; (*HDPts)[minpos].y = tmp;
      tmp = (*HDPts)[i].z; (*HDPts)[i].z = (*HDPts)[minpos].z; (*HDPts)[minpos].z = tmp;
    }
  }

#ifdef TRACE
  printf("Points: \n");
  for(i=0; i < (*numHDPts); i++) {
    printf("%f %f %f - %f\n", (*HDPts)[i].x, (*HDPts)[i].y, (*HDPts)[i].z, adiv[i]);
  }
#endif
  
  //
  // cluster the points
  //
  // Algorithm:
  //	First point creates the first group.
  //	For all the other points:
  //		If the point is close to an existing group
  //			add the point to that group
  //		else
  //			the point starts a new group
  //		endif
  //	endfor
  // end
  //
  
  // initialize data structure
  HDGroup *Groups;
  int numGroups = 0;
  
  if((Groups = new HDGroup[(*numHDPts)]) == NULL) {
    printf("Error allocating memory for working data structures. Abort\n");
    exit(1);
  }
  for(i=0; i < (*numHDPts); i++) {
    if((Groups[i].Points = new int[(*numHDPts)]) == NULL) {
      printf("Error allocating memory for working data structures. Abort\n");
      exit(1);
    }
    Groups[i].numPoints = 0;
  }
  
  bool closeToSomeGroup = false;
  
  // first point creates the first group
  Groups[0].Points[0] = 0;
  Groups[0].numPoints = 1;
  numGroups = 1;
  
  for(i=1; i < (*numHDPts); i++) {
    closeToSomeGroup = false;
    for(j=0; j < numGroups; j++) {
      if(PointIsCloseToGroup(i, j, Groups, HDPts)) {
	// add the point to that group
	Groups[j].Points[Groups[j].numPoints] = i;
	Groups[j].numPoints = Groups[j].numPoints + 1;
	closeToSomeGroup = true;
	break;
      }
    }
    if(!closeToSomeGroup) {
      // start a new group
      Groups[numGroups].Points[0] = i;
      Groups[numGroups].numPoints = 1;
      numGroups++;
    }
  }
  
#ifdef TRACE	
  // print the clustered points:
  printf("Clustered points:\n");
  for(i=0; i < numGroups; i++) {
    printf("%f %f %f\n", 
	   (*HDPts)[Groups[i].Points[0]].x, (*HDPts)[Groups[i].Points[0]].y, (*HDPts)[Groups[i].Points[0]].z);
    for(j=1; j < Groups[i].numPoints; j++) {
      printf("\t%f %f %f\n", 
	     (*HDPts)[Groups[i].Points[j]].x, (*HDPts)[Groups[i].Points[j]].y, (*HDPts)[Groups[i].Points[j]].z);
    }
    
  }
#endif
  
  //
  // Return only the first point in each group as the high divergence points
  //
  
  VoxelPositionDouble* newHDPts;
  
  if((newHDPts = new VoxelPositionDouble[numGroups]) == NULL) {
    printf("GetHighDivergencePoints: UPS! - Error allocating memory for the output array. Abort.\n");
    exit(1);
  }
  
  for(i=0; i < numGroups; i++) {
    newHDPts[i].x = (*HDPts)[Groups[i].Points[0]].x;
    newHDPts[i].y = (*HDPts)[Groups[i].Points[0]].y;
    newHDPts[i].z = (*HDPts)[Groups[i].Points[0]].z;
  }
  
  // delete the old array
  delete [] (*HDPts);
  
  // delete Group data structure
  for(i=0; i < numGroups; i++) {
    delete [] Groups[i].Points;
  }
  delete [] Groups;
  
  // return the new array
  (*HDPts) = newHDPts;
  (*numHDPts) = numGroups;
  
#ifdef TRACE
  printf("Returning points: \n");
  for(i=0; i < (*numHDPts); i++) {
    printf("%f %f %f - %f\n", (*HDPts)[i].x, (*HDPts)[i].y, (*HDPts)[i].z, adiv[i]);
  }
#endif
  
  return true;
}


__device__ __host__ inline Vector interpolation(double x, double y, double z, int sizx, int sizy, int sizz, Vector *forcevec)
{
  float alpha, beta, gamma;
  Vector forceInt;
  long slsz;
  
  alpha=x-int(x);
  beta=y-int(y);
  gamma=z-int(z);
  slsz=sizy*sizx;
  
  forceInt.xd=forcevec[int(z)*slsz + int(y)*sizx + int(x)].xd*(1-alpha)*(1-beta)*(1-gamma)
    +forcevec[(int(z)+1)*slsz + int(y)*sizx + int(x)].xd*(1-alpha)*(1-beta)*gamma
    +forcevec[int(z)*slsz + (int(y)+1)*sizx + int(x)].xd*(1-alpha)*beta*(1-gamma)
    +forcevec[int(z)*slsz + int(y)*sizx + (int(x)+1)].xd*alpha*(1-beta)*(1-gamma)
    +forcevec[(int(z)+1)*slsz + int(y)*sizx + (int(x)+1)].xd*alpha*(1-beta)*gamma
    +forcevec[int(z)*slsz + (int(y)+1)*sizx + (int(x)+1)].xd*alpha*beta*(1-gamma)
    +forcevec[(int(z)+1)*slsz + (int(y)+1)*sizx + int(x)].xd*(1-alpha)*beta*gamma
    +forcevec[(int(z)+1)*slsz + (int(y)+1)*sizx + (int(x)+1)].xd*(alpha*beta*gamma);
  
  forceInt.yd=forcevec[int(z)*slsz + int(y)*sizx + int(x)].yd*(1-alpha)*(1-beta)*(1-gamma)
    +forcevec[(int(z)+1)*slsz + int(y)*sizx + int(x)].yd*(1-alpha)*(1-beta)*gamma
    +forcevec[int(z)*slsz + (int(y)+1)*sizx + int(x)].yd*(1-alpha)*beta*(1-gamma)
    +forcevec[int(z)*slsz + int(y)*sizx + (int(x)+1)].yd*alpha*(1-beta)*(1-gamma)
    +forcevec[(int(z)+1)*slsz + int(y)*sizx + (int(x)+1)].yd*alpha*(1-beta)*gamma
    +forcevec[int(z)*slsz + (int(y)+1)*sizx + (int(x)+1)].yd*alpha*beta*(1-gamma)
    +forcevec[(int(z)+1)*slsz + (int(y)+1)*sizx + int(x)].yd*(1-alpha)*beta*gamma
    +forcevec[(int(z)+1)*slsz + (int(y)+1)*sizx + (int(x)+1)].yd*alpha*beta*gamma;
  
  forceInt.zd=forcevec[int(z)*slsz + int(y)*sizx + int(x)].zd*(1-alpha)*(1-beta)*(1-gamma)
    +forcevec[(int(z)+1)*slsz + int(y)*sizx + int(x)].zd*(1-alpha)*(1-beta)*gamma
    +forcevec[int(z)*slsz + (int(y)+1)*sizx + int(x)].zd*(1-alpha)*beta*(1-gamma)
    +forcevec[int(z)*slsz + int(y)*sizx + (int(x)+1)].zd*alpha*(1-beta)*(1-gamma)
    +forcevec[(int(z)+1)*slsz + int(y)*sizx + (int(x)+1)].zd*alpha*(1-beta)*gamma
    +forcevec[int(z)*slsz + (int(y)+1)*sizx + (int(x)+1)].zd*alpha*beta*(1-gamma)
    +forcevec[(int(z)+1)*slsz + (int(y)+1)*sizx + int(x)].zd*(1-alpha)*beta*gamma
    +forcevec[(int(z)+1)*slsz + (int(y)+1)*sizx + (int(x)+1)].zd*alpha*beta*gamma;
  
  return(forceInt);
}


inline bool PointIsCloseToGroup(int pt, int grp, HDGroup *Groups, VoxelPositionDouble **HDPts) {
  int i;
  for(i=0; i < Groups[grp].numPoints; i++) {
    if(
       (fabs((*HDPts)[pt].x - (*HDPts)[Groups[grp].Points[i]].x) <= 1)	&&
       (fabs((*HDPts)[pt].y - (*HDPts)[Groups[grp].Points[i]].y) <= 1)	&&
       (fabs((*HDPts)[pt].z - (*HDPts)[Groups[grp].Points[i]].z) <= 1))
      {
	return true;
      }
  }
  return false;
}
